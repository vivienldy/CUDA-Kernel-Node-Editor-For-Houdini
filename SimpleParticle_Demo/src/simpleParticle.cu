#include "hip/hip_runtime.h"
#include "simpleParticle.h"
#include "simpleParticle.cuh"
#include "CGUtility.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)
/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char* msg, int line = -1) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        if (line >= 0) {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__global__ void CodeGenerator::CUDAKernel::simpleParticle(glm::vec3 geo1_simpleParticle_parm2_force, float geo1_simpleParticle_parm1_time, glm::vec3* geo1_simpleParticle_geometryvopglobal1_Pbuffer, glm::vec3* geo1_simpleParticle_geometryvopglobal1_vbuffer, float geo1_simpleParticle_geometryvopglobal1_TimeInc, CGGeometry::RAWData geo1_simpleParticle_geometryvopglobal1_OpInput1,  int numThreads)
{
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if(index > numThreads)
    return;

  CodeGenerator::GenericCode::simpleParticle(geo1_simpleParticle_parm2_force, geo1_simpleParticle_parm1_time, geo1_simpleParticle_geometryvopglobal1_Pbuffer, geo1_simpleParticle_geometryvopglobal1_vbuffer, geo1_simpleParticle_geometryvopglobal1_TimeInc, geo1_simpleParticle_geometryvopglobal1_OpInput1,  index);
}

void CodeGenerator::CUDA::simpleParticle (
    glm::vec3 geo1_simpleParticle_parm2_force, float geo1_simpleParticle_parm1_time, CGBuffer<glm::vec3>* geo1_simpleParticle_geometryvopglobal1_Pbuffer, CGBuffer<glm::vec3>* geo1_simpleParticle_geometryvopglobal1_vbuffer, float geo1_simpleParticle_geometryvopglobal1_TimeInc, CGGeometry* geo1_simpleParticle_geometryvopglobal1_OpInput1, 
    int blockSize)
{
    // Buffer malloc
    geo1_simpleParticle_geometryvopglobal1_Pbuffer->malloc();
geo1_simpleParticle_geometryvopglobal1_Pbuffer->loadHostToDevice();

geo1_simpleParticle_geometryvopglobal1_vbuffer->malloc();
geo1_simpleParticle_geometryvopglobal1_vbuffer->loadHostToDevice();

geo1_simpleParticle_geometryvopglobal1_OpInput1->DeviceMalloc();
geo1_simpleParticle_geometryvopglobal1_OpInput1->LoadToDevice();



    // Compute threads num
    int numOfThreads = geo1_simpleParticle_geometryvopglobal1_Pbuffer->getSize();
    // Compute blocks num
    auto num_blocks_threads = ThreadBlockInfo(blockSize, numOfThreads);
    
    // Kernel launch
    CodeGenerator::CUDAKernel::simpleParticle<<<num_blocks_threads.x, num_blocks_threads.y>>>(
        geo1_simpleParticle_parm2_force, geo1_simpleParticle_parm1_time, geo1_simpleParticle_geometryvopglobal1_Pbuffer->getDevicePointer(), geo1_simpleParticle_geometryvopglobal1_vbuffer->getDevicePointer(), geo1_simpleParticle_geometryvopglobal1_TimeInc, geo1_simpleParticle_geometryvopglobal1_OpInput1->GetGeometryRawDataDevice(),  numOfThreads);

    checkCUDAErrorWithLine("simpleParticle error");

    hipDeviceSynchronize();
}